#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream> // Libreria para leer archivos

#include <typeinfo> // for 'typeid' to work

using namespace std;

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, float* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE. line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}

__global__ void kernelTripletF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (hitori[tId-1] == valor)? true : false;
        next = (hitori[tId+1] == valor)? true : false;
        estado[tId] = (back && next) ? 5 : aux;
    }
}

/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable
    6 -> paintable
*/

void checkColAndRow(){




}


void SetHitoriState( int* Hitori, int* Hit_State, int N){

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            int elem = matrix[i + j*N];

            if( elem != 0 )
                // iterar por Fila
                for(int k = 0 + j*N;  k <  ; k++){
                    if(   )



                }
                // iterar por Columna



            else
                continue;

            
        } printf("%d", matrix[i + j*N]);
    }


}



int main(int argc, char* argv[]){

    ifstream FILE;  

    int Hitori;
    int Hit_State;
    int N;
    string line;

    // Abrir el archivo en modo lectura
    
    FILE.open(argv[1], ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line[0]);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        readHitoriFromFile(&FILE, Hitori, N);

        // Ejecutarse Standard Patterns


    }

    FILE.close();



    return 0;
}