#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>  // Libreria para leer archivos
#include <typeinfo> // for 'typeid' to work
#include <tuple>

using namespace std;

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                             Funciones de apoyo                             */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d ", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, int* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE, line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
            
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}


/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable 
    6 -> paintable // Eliminado
*/

/*  
    Función para consistencia del Hitori
    Lo que está función hace es mirar si dos multiples
    en la misma columna o fila tienen el mismo número y si 
    ambos son not paintable (5).
*/
bool isRule4Conform(int* Hit_State, int N){
    
    int i;
    vector<tuple<int, int>> M = getRemainingMultiples(Hit_State, N);
    
    for( i = 0; i < M.size() ; i++){


    }
}

/*  
    Ejecutar cada vez que un multiplo es pintado (6)
     1. Setear todas las celdas adyacentes al múltiplo pintado.
     2. 


*/

bool StandardCyclePattern(int* Hitori, int* Hit_State, int N){




    // Comprueba Regla 4: 
    return isRule4Conform(Hit_State, N);

}

void copyHitoriToHitori(Hit_State, Hit_StateAux, N){
    int i, j;
    for(j = 0; j < N; j++)
        for( i = 0; j < N; j++)
            Hit_StateAux[i +  j*N] = Hit_State[i + j*N];
}

void setNotPaintable(int* Hit_State, tuple<int, int> tup ){
    Hit_State[ get<0>(tup) ] = 5;
}

void paint(int* Hit_State, tuple<int, int> tup){
    Hit_State[ get<0>(tup)] = 6;
    return;
}

// tuple (elem , posElem)
vector<tuple<int , int>> getRemainingMultiples(int* Hit_State, int N){
    
    int i,j;
    int elem;
    int posElem;
    vector<tuple<int, int>> M;
    tuple<int, int> tup;

    /*
        1 -> not multiple
        2 -> multiple per row
        3 -> multiple per column
        4 -> multiple per row and column
        5 -> not paintable 
        6 -> paintable // Eliminado
    */  

    for(j = 0; j < N; j++ ){
        for(i = 0; i < N; i++){
            posElem = i + j*N;
            elem = Hit_State[posElem];
            tup = make_tuple(elem,posElem);
            
            switch(elem) {
                case 2:
                    M.push_back(tup);
                    break;
                case 3:
                    M.push_back(tup);
                    break;
                case 4:
                    M.push_back(tup);
                    break;
                default:
                    break;
            }

        }
    }

    return M;
}



void setInitialHitoriState(int *Hit_State, int N) {

    for(int j = 0; j < N; j++)
    	for(int i = 0; i < N; i++)
    		Hit_State[i + j*N] = 1;    // 1 -> not multiple
    
}

void SetHitoriState( int* Hitori, int* Hit_State, int N){
    
    bool flag1, flag2;

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            flag1 = false; flag2 = false;
            
            int posElem = i + j*N;
            int elem = Hitori[posElem];
            
            // iterar por Fila
            for(int k = j*N;  k < N + j*N ; k++){

                if( k == posElem )
                    continue;

                if( Hitori[k] == elem ){ 
                    flag1 = true;
                    break;
                }    
            }
       
            // iterar por Columna
            for(int t = i; t < N*N ;t += N ){

                if( t == posElem )
                    continue;
                
                if( Hitori[t] == elem){
                    flag2 = true;
                    break;
                }

            }

            if( flag1 == true && flag2 == true) // case 4 -> multiple per row and column
                Hit_State[posElem] = 4;
            else if( flag1 == true )           //2 -> multiple per row 
                Hit_State[posElem] = 2;          
            else if( flag2 == true)            //3 -> multiple per column
                Hit_State[posElem] = 3;  
            
        
        }

    }


}

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                                     CPU                                    */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

void tripletF(int *hitori, int* estado, int N){
    int i, aux;
    bool back, next;
    for(i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if(columna > 0 && columna < N){
            int valor = hitori[i];
            aux = estado[i];
            back = (hitori[i-1] == valor)? true : false;
            next = (hitori[i+1] == vlaor)? true : false;
            estado[i] = (back && next)? 5 : aux;
        }
    }
}

void tripletC(int *hitori, int *estado, int N){
    int i, aux;
    bool up, down;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (fila > 0 && fila < N){
            int valor = hitori[i];
            aux = estado[i];
            up = (hitori[i-N] == valor) ? true : false;
            down = (hitori[i+N] == valor)? true : false;
            estado[i] = (up && down) ? 5 : aux;
        }
    }
}

void rescateF(int *hitori, int *estado, int N){
    int i, aux;
    bool back, next;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (columna > 0 && columna < N){
            int valor = hitori[i];
            aux = estado[i];
            back = (estado[i-1] == 6)? true : false;
            next = (estado[i+1] == 6)? true : false;
            estado[i] = (back || next) ? 5 : aux;
        }
    }
}

void rescateC(int *hitori, int *estado, int N){
    int i, aux;
    bool up, down;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (fila > 0 && fila < N){
            int valor = hitori[i];
            aux = estado[i];
            up = (estado[i-N] == 6)? true : false;
            down = (estado[i+N] == 6)? true : false;
            estado[i] = (up || down) ? 5 : aux;
        }
    }
}

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                         GPU primera implementacion                         */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

__global__ void kernelTripletF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (hitori[tId-1] == valor)? true : false;
        next = (hitori[tId+1] == valor)? true : false;
        estado[tId] = (back && next) ? 5 : aux;
    }
}

__global__ void kernelTripletC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (hitori[tId-N] == valor)? true : false;
        down = (hitori[tId+N] == valor)? true : false;
        estado[tId] = (up && down) ? 5 : aux;
    }
}

__global__ void kernelRescateF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (estado[tId-1] == 6)? true : false;
        next = (estado[tId+1] == 6)? true : false;
        estado[tId] = (back || next) ? 5 : aux;
    }
}


__global__ void kernelRescateC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (estado[tId-N] == 6)? true : false;
        down = (estado[tId+N] == 6)? true : false;
        estado[tId] = (up || down) ? 5 : aux;
    }
}


int main(int argc, char* argv[]){

    fstream FILE;  

    int* Hitori;
    int* Hit_State;
    int N;
    string line;
    vector<tuple<int, int>> M;

    string nameFile = argv[1];
    // Abrir el archivo en modo lectura
    
    FILE.open(nameFile, ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        setInitialHitoriState(Hit_State, N);

        readHitoriFromFile(&FILE, Hitori, N);

        SetHitoriState( Hitori, Hit_State, N);

        /*
        M = getRemainingMultiples(Hit_State, N);

        for( int i = 0; i < M.size() ; i++){

            int poselem = get<1>(M[i]);
            int x = poselem%N;
            int y = poselem/N;
            
            cout << "tuple["<< i <<"] = (" << get<0>(M[i]) <<" , ["<< x << "," <<y <<"] ) " << endl;

        }
        */

        // Parte 1: Ejecutarse Standard Patterns
        /*
        showMatrix(Hitori, N, N);

        printf("\n");
        */

        showMatrix(Hit_State, N, N);
        


        // Parte 2: 
        
        vector<tuple> M; 
        bool flag = false;
        bool inconst;
        int* hitaux;
        int* Hit_StateCpy = new int[N*N];
        
        while(!flag){
            flag = true;
            for( i = 0; i < M.size(); i++ ){
                paint(Hit_State, M[i]);
                // Copia del estado inicial
                copyHitoriToHitori(Hit_State, Hit_StateAux, N);

                inconst = StandardCyclePattern(Hitori, Hit_State, N);

                if( inconst ){
                    
                    // Volver la matrix al estado inicial
                    hit_aux = Hit_State;
                    Hit_State = Hit_StateAux;
                    Hit_StateAux = hit_aux;

                    setNotPaintable(Hitori, Hit_State, N);
                    
                    StandardCyclePattern(Hitori, Hit_State, N);

                    M = getRemainingMultiples(Hit_State, N);

                    flag = false;
                    
                    break; // seteo i = 0

                }

                

            }




        }
        





    }

    FILE.close();



    return 0;
}