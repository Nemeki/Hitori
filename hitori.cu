#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>  // Libreria para leer archivos
#include <typeinfo> // for 'typeid' to work

using namespace std;

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d ", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, int* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE, line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
            
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}

/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable
    6 -> paintable
*/

void setInitialHitoriState(int *Hit_State, int N) {

    for(int j = 0; j < N; j++)
    	for(int i = 0; i < N; i++)
    		Hit_State[i + j*N] = 1;    // 1 -> not multiple
    
}

void SetHitoriState( int* Hitori, int* Hit_State, int N){
    bool flag1, flag2;

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            flag1 = false; flag2 = false;
            
            int posElem = i + j*N;
            int elem = Hitori[posElem];
            
            // iterar por Fila
            for(int k = j*N;  k < N + j*N ; k++){

                if( k == posElem )
                    continue;

                if( Hitori[k] == elem ){ 
                    flag1 = true;
                    break;
                }    
            }
       
            // iterar por Columna
            for(int t = i; t < N*N ;t += N ){

                if( t == posElem )
                    continue;
                
                if( Hitori[t] == elem){
                    flag2 = true;
                    break;
                }

            }

            if( flag1 == true && flag2 == true) // case 4 -> multiple per row and column
                Hit_State[posElem] = 4;
            else if( flag1 == true )           //2 -> multiple per row 
                Hit_State[posElem] = 2;          
            else if( flag2 == true)            //3 -> multiple per column
                Hit_State[posElem] = 3;  
            
        
        }

    }


}


int main(int argc, char* argv[]){

    fstream FILE;  

    int* Hitori;
    int* Hit_State;
    int N;
    string line;

    string nameFile = argv[1];
    // Abrir el archivo en modo lectura
    
    FILE.open(nameFile, ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        setInitialHitoriState(Hit_State, N);

        readHitoriFromFile(&FILE, Hitori, N);

        SetHitoriState( Hitori, Hit_State, N);

        // Ejecutarse Standard Patterns
        showMatrix(Hitori, N, N);

        printf("\n");

        showMatrix(Hit_State, N, N);


    }

    FILE.close();

    return 0;
}