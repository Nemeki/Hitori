#include <iostream>
#include <stdio.h>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream> // Libreria para leer archivos

#include <typeinfo> // for 'typeid' to work

using namespace std;

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, float* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE. line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}

/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable
    6 -> paintable
*/

void checkColAndRow(){




}


void SetHitoriState( int* Hitori, int* Hit_State, int N){

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            int elem = matrix[i + j*N];

            if( elem != 0 )
                // iterar por Fila
                for(int k = 0 + j*N;  k <  ; k++){
                    if(   )



                }
                // iterar por Columna



            else
                continue;

            
        } printf("%d", matrix[i + j*N]);
    }


}



int main(int argc, char* argv[]){

    ifstream FILE;  

    int Hitori;
    int Hit_State;
    int N;
    string line;

    // Abrir el archivo en modo lectura
    
    FILE.open(argv[1], ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line[0]);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        readHitoriFromFile(&FILE, Hitori, N);

        // Ejecutarse Standard Patterns


    }

    FILE.close();

    return 0;
}