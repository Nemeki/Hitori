#include <iostream>
#include <stdio.h>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream> // Libreria para leer archivos

#include <typeinfo> // for 'typeid' to work

using namespace std;

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, float* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE. line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}


int main(int argc, char* argv[]){

    ifstream FILE;  

    int Hitori;
    int Hit_State;
    int N;
    string line;

    // Abrir el archivo en modo lectura
    
    FILE.open(argv[1], ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line[0]);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        readHitoriFromFile(&FILE, Hitori, N);


    }

    FILE.close();

    return 0;
}