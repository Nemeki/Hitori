#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>  // Libreria para leer archivos
#include <typeinfo> // for 'typeid' to work

using namespace std;

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d ", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void readHitoriFromFile(fstream* FILE, int* matrixH, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE, line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixH[j++] = stoi(row[i]);
            
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}

__global__ void kernelTripletF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (hitori[tId-1] == valor)? true : false;
        next = (hitori[tId+1] == valor)? true : false;
        estado[tId] = (back && next) ? 5 : aux;
    }
}

__global__ void kernelTripletC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (hitori[tId-N] == valor)? true : false;
        down = (hitori[tId+N] == valor)? true : false;
        estado[tId] = (up && down) ? 5 : aux;
    }
}

__global__ void kernelRescateF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (estado[tId-1] == 6)? true : false;
        next = (estado[tId+1] == 6)? true : false;
        estado[tId] = (back || next) ? 5 : aux;
    }
}

__global__ void kernelRescateC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (estado[tId-N] == 6)? true : false;
        down = (estado[tId+N] == 6)? true : false;
        estado[tId] = (up || down) ? 5 : aux;
    }
}

/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable 
    6 -> paintable // Eliminado
*/

void setInitialHitoriState(int *Hit_State, int N) {

    for(int j = 0; j < N; j++)
    	for(int i = 0; i < N; i++)
    		Hit_State[i + j*N] = 1;    // 1 -> not multiple
    
}

void SetHitoriState( int* Hitori, int* Hit_State, int N){
    bool flag1, flag2;

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            flag1 = false; flag2 = false;
            
            int posElem = i + j*N;
            int elem = Hitori[posElem];
            
            // iterar por Fila
            for(int k = j*N;  k < N + j*N ; k++){

                if( k == posElem )
                    continue;

                if( Hitori[k] == elem ){ 
                    flag1 = true;
                    break;
                }    
            }
       
            // iterar por Columna
            for(int t = i; t < N*N ;t += N ){

                if( t == posElem )
                    continue;
                
                if( Hitori[t] == elem){
                    flag2 = true;
                    break;
                }

            }

            if( flag1 == true && flag2 == true) // case 4 -> multiple per row and column
                Hit_State[posElem] = 4;
            else if( flag1 == true )           //2 -> multiple per row 
                Hit_State[posElem] = 2;          
            else if( flag2 == true)            //3 -> multiple per column
                Hit_State[posElem] = 3;  
            
        
        }

    }


}


int main(int argc, char* argv[]){

    fstream FILE;  

    int* Hitori;
    int* Hit_State;
    int N;
    string line;

    string nameFile = argv[1];
    // Abrir el archivo en modo lectura
    
    FILE.open(nameFile, ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];

        setInitialHitoriState(Hit_State, N);

        readHitoriFromFile(&FILE, Hitori, N);

        SetHitoriState( Hitori, Hit_State, N);

        // Ejecutarse Standard Patterns
        showMatrix(Hitori, N, N);

        printf("\n");

        showMatrix(Hit_State, N, N);


    }

    FILE.close();



    return 0;
}