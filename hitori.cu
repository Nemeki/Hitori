#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>  // Libreria para leer archivos
#include <typeinfo> // for 'typeid' to work
#include <tuple>

using namespace std;

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                             Funciones de apoyo                             */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

__constant__ int HitoriCM[5*5];  //FIXME: Cambiar cuando se actualice N y M

// Función para Splitear un String
void tokenize(string const &str, const char delim, vector<string> &out) {
    // construct a stream from the string
    stringstream ss(str);
    string s;

    while (getline(ss, s, delim)) {
        out.push_back(s);
    }

    return;

}

/*
 *  Impresor de Matrix 2D - Almacenada linealmente
 */
void showMatrix(int *matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
    		printf("%d ", matrix[i + j*N]);
    	printf("\n");
    }
    printf("\n");
}

void showMatrix(string* matrix, int N, int M) {
    for(int j = 0; j < M; j++){
    	for(int i = 0; i < N; i++)
            cout << matrix[i + j*N] << " ";
    	printf("\n");
    }
    printf("\n");
}


void readHitoriFromFile(fstream* FILE, int* matrixH, string* matrixHstr, int N){

    int i, j = 0;

    const char delim = ' ';

    string line;
    vector<string> row;
    
    while( getline(*FILE, line)){

        tokenize(line, delim, row);

        for(i = 0; i < N ; i++){
            matrixHstr[j] = row[i];
            matrixH[j++] = stoi(row[i]);        
        }
        // Limpiar el buffer de salida  
        row.clear();
    }

}


/*
    1 -> not multiple
    2 -> multiple per row
    3 -> multiple per column
    4 -> multiple per row and column
    5 -> not paintable 
    6 -> paintable // Eliminado
*/

// tuple (elem , posElem)
vector<tuple<int , int>> getRemainingMultiples(int* Hit_State, int N){
    
    int i,j;
    int elem;
    int posElem;
    vector<tuple<int, int>> M;
    tuple<int, int> tup;

    /*
        1 -> not multiple
        2 -> multiple per row
        3 -> multiple per column
        4 -> multiple per row and column
        5 -> not paintable 
        6 -> paintable // Eliminado
    */  

    for(j = 0; j < N; j++ ){
        for(i = 0; i < N; i++){
            posElem = i + j*N;
            elem = Hit_State[posElem];
            tup = make_tuple(elem,posElem);
            
            switch(elem) {
                case 2:
                    M.push_back(tup);
                    break;
                case 3:
                    M.push_back(tup);
                    break;
                case 4:
                    M.push_back(tup);
                    break;
                default:
                    break;
            }

        }
    }

    return M;
}
/*  
    Función para consistencia del Hitori
    Lo que está función hace es mirar si dos multiples
    en la misma columna o fila tienen el mismo número y si 
    ambos son not paintable (5).
*/
bool isRule4Conform(int* Hit_State, int N){
    
    int i;
    vector<tuple<int, int>> M = getRemainingMultiples(Hit_State, N);
    
    for( i = 0; i < M.size() ; i++){


    }

    return true;
}

/*  
    Ejecutar cada vez que un multiplo es pintado (6)
     1. Setear todas las celdas adyacentes al múltiplo pintado.
     2. 


*/

bool StandardCyclePattern(int* Hitori, int* Hit_State, int N){

    // Comprueba Regla 4: 
    // return isRule4Conform(Hit_State, N);

    return true;

}

void copyHitoriToHitori(int* Hit_State, int* Hit_StateAux, int N){
    int i, j;
    for(j = 0; j < N; j++)
        for( i = 0; j < N; j++)
            Hit_StateAux[i +  j*N] = Hit_State[i + j*N];
}

void setNotPaintable(int* Hit_State, tuple<int, int> tup ){
    Hit_State[ get<0>(tup) ] = 5;
}

void paint(int* Hit_State, tuple<int, int> tup){
    Hit_State[ get<0>(tup)] = 6;
    return;
}



void setInitialHitoriState(int *Hit_State, int N) {

    for(int j = 0; j < N; j++)
    	for(int i = 0; i < N; i++)
    		Hit_State[i + j*N] = 1;    // 1 -> not multiple
    
}

void SetHitoriState( int* Hitori, int* Hit_State, int N){
    
    bool flag1, flag2;

    for(int j = 0; j < N; j++){
    	for(int i = 0; i < N; i++){
            
            flag1 = false; flag2 = false;
            
            int posElem = i + j*N;
            int elem = Hitori[posElem];
            
            // iterar por Fila
            for(int k = j*N;  k < N + j*N ; k++){

                if( k == posElem )
                    continue;

                if( Hitori[k] == elem ){ 
                    flag1 = true;
                    break;
                }    
            }
       
            // iterar por Columna
            for(int t = i; t < N*N ;t += N ){

                if( t == posElem )
                    continue;
                
                if( Hitori[t] == elem){
                    flag2 = true;
                    break;
                }

            }

            if( flag1 == true && flag2 == true) // case 4 -> multiple per row and column
                Hit_State[posElem] = 4;
            else if( flag1 == true )           //2 -> multiple per row 
                Hit_State[posElem] = 2;          
            else if( flag2 == true)            //3 -> multiple per column
                Hit_State[posElem] = 3;
        }

    }


}

void updateHitori(string* Hitori_Str, int* Hit_State, int N){
    int i, j;

    for( j = 0; j < N; j++){
        for( i = 0; i < N; i++){
            if( Hit_State[i + j*N] == 6)
                Hitori_Str[i + j*N] = "X";
        }
    }
    return;
}
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                                     CPU                                    */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

void tripletF(int *hitori, int* estado, int N){
    int i, aux;
    bool back, next;
    for(i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if(columna > 0 && columna < N){
            int valor = hitori[i];
            aux = estado[i];
            back = (hitori[i-1] == valor)? true : false;
            next = (hitori[i+1] == valor)? true : false;
            estado[i] = (back && next)? 5 : aux;
        }
    }
}

void tripletC(int *hitori, int *estado, int N){
    int i, aux;
    bool up, down;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (fila > 0 && fila < N){
            int valor = hitori[i];
            aux = estado[i];
            up = (hitori[i-N] == valor) ? true : false;
            down = (hitori[i+N] == valor)? true : false;
            estado[i] = (up && down) ? 5 : aux;
        }
    }
}

void rescateF(int *hitori, int *estado, int N){
    int i, aux;
    bool back, next;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (columna > 0 && columna < N){
            int valor = hitori[i];
            aux = estado[i];
            back = (estado[i-1] == 6)? true : false;
            next = (estado[i+1] == 6)? true : false;
            estado[i] = (back || next) ? 5 : aux;
        }
    }
}

void rescateC(int *hitori, int *estado, int N){
    int i, aux;
    bool up, down;
    for (i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        if (fila > 0 && fila < N){
            int valor = hitori[i];
            aux = estado[i];
            up = (estado[i-N] == 6)? true : false;
            down = (estado[i+N] == 6)? true : false;
            estado[i] = (up || down) ? 5 : aux;
        }
    }
}

void DobleC(int* hitori,int *estado, int N){

    int f; //Fila en que esta
	int c; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    for(int i = 0; i < N*N; i++) {
        f = i / N;
        c = i % N;
        int valor = hitori[i];
        for(int j = 0; j < N; j++){
            pos = c+N*j;
            doble = (ant && i != pos && hitori[pos] == valor)? true : doble;
            ant = (i != pos && hitori[pos] == valor)? true : false;
        }
        if(doble) {
            estado[i] = 6;
        }
    }

}

void DobleF(int* hitori,int *estado, int N){
    
    int f; //Fila en que esta
	int c; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    for(int i = 0; i < N*N; i++) {
        f = i / N;
        c = i % N;
        int valor = hitori[i];
        for(int j = 0; j < N; j++){
            pos = f+j;
            doble = (ant && i != pos && hitori[pos] == valor)? true : doble;
            ant = (i != pos && hitori[pos] == valor)? true : false;
        }
        if(doble) {
            estado[i] = 6;
        }
    }
}

void muerteF(int *hitori, int *estado, int N){
    int i, aux1, aux2;
    for(i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        int valor = hitori[i];
        aux1 = estado[i];
        if(aux1 != 5 && aux1 !=6){
            for(int j = 0; j < N; j++){
                aux2 = hitori[fila + j];
                if(valor == aux2) aux1 = (estado[fila+j] == 5)? 6 : aux1;
            }
        }
    }
}

void muerteC(int *hitori, int *estado, int N){
    int i, aux1, aux2;
    for(i = 0; i < N*N; i++){
        int fila = i/N;
        int columna = i%N;
        int valor = hitori[i];
        aux1 = estado[i];
        if(aux1 != 5 && aux1 !=6){
            for(int j = 0; j < N; j++){
                aux2 = hitori[columna + N*j];
                if(valor == aux2) aux1 = (estado[columna + N*j] == 5)? 6 : aux1;
            }
        }
    }
}

void funcionCPU(int* Hitori, int* estado, int N){

    int i;
    // Ejecutar patrones 
    tripletF(Hitori, estado, N);
    tripletC(Hitori, estado, N);
    //DobleF(Hitori, estado, N);
    //DobleC(Hitori, estado, N);
 
    for(i = 0; i < 10; i++){
        muerteF(Hitori, estado, N);
        muerteC(Hitori, estado, N);
        rescateC(Hitori, estado, N);
        rescateF(Hitori, estado, N);
    }

    return;

}

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                         GPU primera implementacion                         */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

/* -------------------------- Deteccion de patrones ------------------------- */

__global__ void kernelTripletF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (hitori[tId-1] == valor)? true : false;
        next = (hitori[tId+1] == valor)? true : false;
        estado[tId] = (back && next) ? 5 : aux;
    }
}

__global__ void kernelTripletC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (hitori[tId-N] == valor)? true : false;
        down = (hitori[tId+N] == valor)? true : false;
        estado[tId] = (up && down) ? 5 : aux;
    }

}

__global__ void kernelDobleF(int *hitori, int *estado, int N){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    if(tId < N*N) {
        int valor = hitori[tId];
        for(int i = 0; i < N; i++){
            pos = f+i;
            doble = (ant && tId != pos && hitori[pos] == valor)? true : doble;
            ant = (tId != pos && hitori[pos] == valor)? true : false;
        }
        if(doble) {
            estado[tId] = 6;
        }
    }
}

__global__ void kernelDobleC(int *hitori, int *estado, int N){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    if(tId < N*N) {
        int valor = hitori[tId];
        for(int i = 0; i < N; i++){
            pos = c+N*i;
            doble = (ant && tId != pos && hitori[pos] == valor)? true : doble;
            ant = (tId != pos && hitori[pos] == valor)? true : false;
        }
        if(doble) {
            estado[tId] = 6;
        }
    }
}

/* ---------------------------- Funciones del for --------------------------- */

__global__ void kernelRescateF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        back = (estado[tId-1] == 6)? true : false;
        next = (estado[tId+1] == 6)? true : false;
        estado[tId] = (back || next) ? 5 : aux;
    }

}


__global__ void kernelRescateC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = hitori[tId];
        aux = estado[tId];
        up = (estado[tId-N] == 6)? true : false;
        down = (estado[tId+N] == 6)? true : false;
        estado[tId] = (up || down) ? 5 : aux;
    }
}

__global__ void kernelMuerteF(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int aux1, aux2, aux3;

    if(tId < N*N) {
        int valor = hitori[tId];
        aux1 = estado[tId];
        if(aux1 != 5 && aux1 != 6){
            for(int i = 0; i < N; i++){
                aux2 = hitori[f+i];
                if(valor == aux2){
                    aux1 = (estado[f+i] == 5)? 6 : aux1;
                }
            }
            estado[tId] = aux1;
        }
    }

}

__global__ void kernelMuerteC(int *hitori, int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int aux1, aux2, aux3;

    if(tId < N*N) {
        int valor = hitori[tId];
        aux1 = estado[tId];
        if (aux1 != 5 && aux1 != 6){
            for(int i = 0; i < N; i++){
                aux2 = hitori[c+N*i];
                if(valor == aux2){
                    aux1 = (estado[c+N*i] == 5)? 6 : aux1;
                }
            }
            estado[tId] = aux1;
        }
    }
}

/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                         GPU segunda implementacion                         */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

__global__ void kernelTripletF_CM(int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = HitoriCM[tId];
        aux = estado[tId];
        back = (HitoriCM[tId-1] == valor)? true : false;
        next = (HitoriCM[tId+1] == valor)? true : false;
        estado[tId] = (back && next) ? 5 : aux;
    }

}

__global__ void kernelTripletC_CM(int *estado, int N){

    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = HitoriCM[tId];
        aux = estado[tId];
        up = (HitoriCM[tId-N] == valor)? true : false;
        down = (HitoriCM[tId+N] == valor)? true : false;
        estado[tId] = (up && down) ? 5 : aux;
    }

}

__global__ void kernelRescateF_CM(int *estado, int N){

    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool back, next;
    int aux;

    if(tId < N*N && c > 0 && c < N) {
        int valor = HitoriCM[tId];
        aux = estado[tId];
        back = (estado[tId-1] == 6)? true : false;
        next = (estado[tId+1] == 6)? true : false;
        estado[tId] = (back || next) ? 5 : aux;
    }

}

__global__ void kernelRescateC_CM(int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool up, down;
    int aux;

    if(tId < N*N && f > 0 && f < N) {
        int valor = HitoriCM[tId];
        aux = estado[tId];
        up = (estado[tId-N] == 6)? true : false;
        down = (estado[tId+N] == 6)? true : false;
        estado[tId] = (up || down) ? 5 : aux;
    }
}

__global__ void kernelDobleC_CM(int *estado, int N){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    if(tId < N*N) {
        int valor = HitoriCM[tId];
        for(int i = 0; i < N; i++){
            pos = c+N*i;
            doble = (ant && tId != pos && HitoriCM[pos] == valor)? true : doble;
            ant = (tId != pos && HitoriCM[pos] == valor)? true : false;
        }
        if(doble) {
            estado[tId] = 6;
        }
    }
}

__global__ void kernelDobleF_CM(int *estado, int N){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    bool ant = false;
    bool doble = false;
    int pos;

    if(tId < N*N) {
        int valor = HitoriCM[tId];
        for(int i = 0; i < N; i++){
            pos = f+i;
            doble = (ant && tId != pos && HitoriCM[pos] == valor)? true : doble;
            ant = (tId != pos && HitoriCM[pos] == valor)? true : false;
        }
        if(doble) {
            estado[tId] = 6;
        }
    }
}

__global__ void kernelMuerteF_CM(int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int aux1, aux2, aux3;

    if(tId < N*N) {
        int valor = HitoriCM[tId];
        aux1 = estado[tId];
        if(aux1 != 5 && aux1 != 6){
            for(int i = 0; i < N; i++){
                aux2 = HitoriCM[f+i];
                if(valor == aux2){
                    aux1 = (estado[f+i] == 5)? 6 : aux1;
                }
            }
            estado[tId] = aux1;
        }
    }

}

__global__ void kernelMuerteC_CM(int *estado, int N){
	
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int f = tId / N; //Fila en que esta
	int c = tId % N; //Columna en la que esta
    int aux1, aux2, aux3;

    if(tId < N*N) {
        int valor = HitoriCM[tId];
        aux1 = estado[tId];
        if (aux1 != 5 && aux1 != 6){
            for(int i = 0; i < N; i++){
                aux2 = HitoriCM[c+N*i];
                if(valor == aux2){
                    aux1 = (estado[c+N*i] == 5)? 6 : aux1;
                }
            }
            estado[tId] = aux1;
        }
    }
}


/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */
/*                                    Main                                    */
/* ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ */

int main(int argc, char* argv[]){

    fstream FILE;  

    int* Hitori;
    string* Hitori_Str;
    int* Hit_State; 
     
    int N;
    string line;
    vector<tuple<int, int>> M;

    string nameFile = argv[1];
    // Abrir el archivo en modo lectura
    
    FILE.open(nameFile, ios::in);

    if(!FILE){
        cerr << "Unable to open file!" << endl;
        exit(1);
    }

    if( FILE.is_open() ){

        getline(FILE, line);
        
        N = stoi(line);

        Hitori = new int[N*N];
        Hit_State = new int[N*N];
        Hitori_Str = new string[N*N];

        setInitialHitoriState(Hit_State, N);

        readHitoriFromFile(&FILE, Hitori, Hitori_Str, N);

        SetHitoriState( Hitori, Hit_State, N);

        // Parte CPU
        // Inicialización variables de tiempo
        clock_t t1, t2;
        double ms; 

        t1 = clock();
        funcionCPU(Hitori, Hit_State, N);
        t2 = clock();
        ms = 1000.0 * (double)(t2 - t1) / CLOCKS_PER_SEC;   
        printf("Tiempo de CPU: %5f \n", ms);
        //cout << "Tiempo CPU: " << ms << "[ms]" << endl;

                     
        // Visualizar Hitori
        updateHitori(Hitori_Str, Hit_State, N);
        showMatrix(Hitori_Str, N, N);
        printf("\n Hitori Estado \n");
        showMatrix(Hit_State, N, N); 

        SetHitoriState( Hitori, Hit_State, N);

        // Parte GPU 1 
        // Def tiempos GPU
        int* HitoriDev, *Hit_StateDev;
        hipEvent_t ct1, ct2;
        float dt;
        hipEventCreate(&ct1);
        hipEventCreate(&ct2);

        int block_size = 256;					 		              // múltiplo de 32
        int grid_size  = (int)ceil((float)(N*N)/block_size);          // ceil : función techo 

        hipMalloc(&HitoriDev, sizeof(int)*N*N);
        hipMalloc(&Hit_StateDev, sizeof(int)*N*N);

        hipEventCreate(&ct1);
        hipEventCreate(&ct2);
        hipEventRecord(ct1);
        hipMemcpy(HitoriDev, Hitori, N*N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(Hit_StateDev, Hit_State, N*N*sizeof(int), hipMemcpyHostToDevice);
        kernelTripletF<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
        kernelTripletC<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
        //kernelDobleF<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
        //kernelDobleC<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
        for(int i = 0; i < 10; i++){
            kernelMuerteF<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
            kernelMuerteC<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
            kernelRescateF<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
            kernelRescateC<<<grid_size, block_size>>>(HitoriDev, Hit_StateDev, N);
        }
        hipMemcpy(Hit_State, Hit_StateDev, N*N*sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(ct2);
        hipEventSynchronize(ct2);
        hipEventElapsedTime(&dt, ct1, ct2);

        cout << "Tiempo GPU 1: " << dt << "[ms]" << endl;

                
        // Visualizar Hitori
        updateHitori(Hitori_Str, Hit_State, N);
        showMatrix(Hitori_Str, N, N);
        printf("\n Hitori Estado \n");
        showMatrix(Hit_State, N, N); 


        SetHitoriState( Hitori, Hit_State, N);

        // Parte GPU 2
        int* Hit_StateDev2;
        hipMalloc(&Hit_StateDev2, sizeof(int)*N*N);
    
        hipEventRecord(ct1);
        hipMemcpyToSymbol(HIP_SYMBOL(HitoriCM), Hitori, N*N*sizeof(int), 0, hipMemcpyHostToDevice); // Para kernel CM
        hipMemcpy(Hit_StateDev2, Hit_State, N*N*sizeof(int), hipMemcpyHostToDevice);
        kernelTripletF_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
        kernelTripletC_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
        //kernelDobleF_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
        //kernelDobleC_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
        for(int i = 0; i < 10; i++){
            kernelMuerteF_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
            kernelMuerteC_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
            kernelRescateF_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
            kernelRescateC_CM<<<grid_size, block_size>>>(Hit_StateDev2, N);
        }
        hipMemcpy(Hit_State, Hit_StateDev2, N*N*sizeof(int), hipMemcpyDeviceToHost);
        hipEventRecord(ct2);
        hipEventSynchronize(ct2);
        hipEventElapsedTime(&dt, ct1, ct2);


        cout << "Tiempo GPU 2: " << dt << "[ms]" << endl;
                     
        // Visualizar Hitori
        updateHitori(Hitori_Str, Hit_State, N);
        showMatrix(Hitori_Str, N, N);
        printf("\n Hitori Estado \n");
        showMatrix(Hit_State, N, N); 

        /*
        M = getRemainingMultiples(Hit_State, N);

        for( int i = 0; i < M.size() ; i++){

            int poselem = get<1>(M[i]);
            int x = poselem%N;
            int y = poselem/N;
            
            cout << "tuple["<< i <<"] = (" << get<0>(M[i]) <<" , ["<< x << "," <<y <<"] ) " << endl;

        }
        */

        // Parte 1: Ejecutarse Standard Patterns    
    
        
        // Parte 2: 
        
        /*
        vector<tuple> M; 
        bool flag = false;
        bool inconst;
        int* hitaux;
        int* Hit_StateCpy = new int[N*N];
        
        while(!flag){
            flag = true;
            for( i = 0; i < M.size(); i++ ){
                paint(Hit_State, M[i]);
                // Copia del estado inicial
                copyHitoriToHitori(Hit_State, Hit_StateAux, N);

                inconst = StandardCyclePattern(Hitori, Hit_State, N);

                if( inconst ){
                    
                    // Volver la matrix al estado inicial
                    hit_aux = Hit_State;
                    Hit_State = Hit_StateAux;
                    Hit_StateAux = hit_aux;

                    setNotPaintable(Hitori, Hit_State, N);
                    
                    StandardCyclePattern(Hitori, Hit_State, N);

                    M = getRemainingMultiples(Hit_State, N);

                    flag = false;
                    
                    break; // seteo i = 0

                }

                

            }




        }
        */

    }

    FILE.close();



    return 0;
}