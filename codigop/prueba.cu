#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <fstream>  // Libreria para leer archivos
#include <typeinfo> // for 'typeid' to work
#include <tuple> 
using namespace std;

// tuple (elem , posElem)
vector<tuple<int , int>> getRemainingMultiples(int* Hit_State,int N){
    
    int i,j;
    int elem;
    int posElem;
    vector<tuple<int, int>> M;
    tuple<int, int> tup;

    /*
        1 -> not multiple
        2 -> multiple per row
        3 -> multiple per column
        4 -> multiple per row and column
        5 -> not paintable 
        6 -> paintable // Eliminado
    */  

    for(i = 0; i < N; i++ ){
        for(j = 0; j < N; j++){
            posElem = i + j*N;
            elem = Hit_State[posElem];
            tup = make_tuple(elem,posElem);
            
            switch(elem) {
                case 2:
                    M.push_back(tup);
                    break;
                case 3:
                    M.push_back(tup);
                    break;
                case 4:
                    M.push_back(tup);
                    break;
                default:
                    break;
            }

        }
    }

    return M;
}


int main( ){

    tuple<int, int> tup1, tup2;

    tup1 = make_tuple(1,3);

    tup2 = make_tuple(1,5);

    vector<tuple<int, int>> M;
    
    M.push_back(tup1);
    M.push_back(tup2);

    /*
    for( int i = 0; i < M.size() ; i++){
        
        cout << "tuple["<< i <<"] = (" << get<0>(M[i]) <<" ," << get<1>(M[i]) << ") " << endl;

    }
    */
    int N = 10;
    string* Hitori = new string[N*N];

    Hitori[0] = "dsadadasdas";

    cout << "EL VALOR DE GIROTIRO : "<< Hitori[0] << endl;


    return 0;

}